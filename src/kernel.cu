#include "hip/hip_runtime.h"
template<int NUM_LIMBS>
struct BigInt {
    unsigned long long limbs[NUM_LIMBS];

    __device__ BigInt() {
        for (int i = 0; i < NUM_LIMBS; i++) {
            limbs[i] = 0;
        }
    }

    __device__ BigInt(const unsigned long long* input_limbs) {
        for (int i = 0; i < NUM_LIMBS; i++) {
            limbs[i] = input_limbs[i];
        }
    }

    __device__ BigInt operator^(const BigInt& other) const {
        BigInt result;
        for (int i = 0; i < NUM_LIMBS; i++) {
            result.limbs[i] = this->limbs[i] ^ other.limbs[i];
        }
        return result;
    }

    __device__ BigInt operator&(const BigInt& other) const {
        BigInt result;
        for (int i = 0; i < NUM_LIMBS; i++) {
            result.limbs[i] = this->limbs[i] & other.limbs[i];
        }
        return result;
    }

    __device__ void atomic_xor_into(BigInt& target) const {
        for (int i = 0; i < NUM_LIMBS; i++) {
            atomicXor(&target.limbs[i], this->limbs[i]);
        }
    }
};

template<int NUM_LIMBS>
__global__ void masked_xor(BigInt<NUM_LIMBS>* a, BigInt<NUM_LIMBS>* b, BigInt<NUM_LIMBS>* c, unsigned int num_elements) {
    int idx = (blockIdx.x * blockDim.x + threadIdx.x);
    BigInt<NUM_LIMBS> local_result;
    
    if (idx < num_elements) {
        BigInt<NUM_LIMBS> and_result = a[idx] & b[idx];
        local_result = local_result ^ and_result;
    }
    
    extern __shared__ unsigned char shared_mem[];
    BigInt<NUM_LIMBS>* shared_data = reinterpret_cast<BigInt<NUM_LIMBS>*>(shared_mem);
    shared_data[threadIdx.x] = local_result;
    
    __syncthreads();
    
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shared_data[threadIdx.x] = shared_data[threadIdx.x] ^ shared_data[threadIdx.x + stride];
        }
        __syncthreads();
    }
    
    if (threadIdx.x == 0) {
        shared_data[0].atomic_xor_into(c[0]);
    }
}

extern "C" __global__ void masked_xor_64(BigInt<1>* a, BigInt<1>* b, BigInt<1>* c, unsigned int num_elements) {
    masked_xor<1>(a, b, c, num_elements);
}

extern "C" __global__ void masked_xor_128(BigInt<2>* a, BigInt<2>* b, BigInt<2>* c, unsigned int num_elements) {
    masked_xor<2>(a, b, c, num_elements);
}

extern "C" __global__ void masked_xor_256(BigInt<4>* a, BigInt<4>* b, BigInt<4>* c, unsigned int num_elements) {
    masked_xor<4>(a, b, c, num_elements);
}
